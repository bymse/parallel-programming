#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
using namespace std::chrono;

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 500

__global__ void gaussian_elimination(float* A, int pivot)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N)
    {
        for (int i = pivot + 1; i < N; i++)
        {
            float factor = A[i * N + pivot] / A[pivot * N + pivot];
            for (int j = pivot; j < N; j++)
            {
                A[i * N + j] -= factor * A[pivot * N + j];
            }
        }
    }
}

int main()
{
    float *A = new float[N * N];
    float det = 1.0;

    for (int i = 0; i < N * N; i++)
    {
        A[i] = rand() % 100;
    }


    float* d_A;
    hipMalloc((void**)&d_A, N * N * sizeof(float));

    // Copy data to device memory
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);

    auto start = high_resolution_clock::now();
    // Gaussian elimination
    for (int i = 0; i < N; i++)
    {
        gaussian_elimination << <1, N >> > (d_A, i);
    }

    // Copy data back to host memory
    hipMemcpy(A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate determinant
    for (int i = 0; i < N; i++)
    {
        det *= A[i * N + i];
    }

    auto stop = high_resolution_clock::now();

    hipFree(d_A);

    std::cout << "Determinant: " << det << std::endl;

   
    auto duration = duration_cast<seconds>(stop - start).count();

    std::cout << "Seconds: " << duration << std::endl;

    delete[] A;

    return 0;
}